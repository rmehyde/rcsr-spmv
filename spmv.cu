#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <limits.h>
#include <omp.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "hipsparse.h"
#include "serial_tools.h"

// define constants
#define VERBOSE 1
#define PRINT_ARRS 1 // print the dense arrays
// DO NOT GO CHANING THIS SHIT WILLY NILLY!
// ELLs must be coalesced according to block size!
#define BLOCKSIZE 32
#define SHAREDMEM 12288

struct sysinfo {
	int numdevices; // number of devices in the system
	int sharedsize; // minimum constant memory size in bytes of any device on the system
	int warpsize; // minimum warpsize in threads of any device on the system
};

char * logbuf;
struct timespec wallstart;
clock_t cpustart;

__global__ void ComputeRibbon(int * rowsts, int * cols, float * vals, float * d_xslice, int ribwidth, int ribid, int fullribwidth, int * row_ranges, float * ribres) {
	int thread = threadIdx.x;
	// slices should be padded to warp size
	// ******** FIX IT TO BLOCK SIZE ********
	__shared__ float x_slice[SHAREDMEM];
	int max_iters = ribwidth/blockDim.x;
	if(ribwidth%blockDim.x != 0)
		max_iters++;
	for(int i=0; i<max_iters; i++) {
//		printf("moving index %d of slice to shared memory (max_iters = %d)\n", i*blockDim.x+thread, max_iters);
		x_slice[i*blockDim.x+thread] = d_xslice[i*blockDim.x+thread];
	}
	__syncthreads();

	// compute the range of rows assigned to this thread
	int startrow = row_ranges[thread];
	int endrow = row_ranges[thread+1];

	double rowsum;
	for(int row=startrow; row < endrow; row++) {
		rowsum = 0.0f;
		for(int s = rowsts[row]; s < rowsts[row+1]; s++) {
			rowsum += x_slice[cols[s]]*vals[s];
		}
		ribres[row] = (float)rowsum;
	}
}

// trivial reductive sum
__global__ void VVSum(float * a, float * b) {
	a[blockIdx.x*BLOCKSIZE+threadIdx.x] += b[blockIdx.x*BLOCKSIZE+threadIdx.x];
}

int ** build_row_ranges(struct ricsr mtx, int blocksize) {
	int ** row_ranges = (int **)malloc(mtx.numrib * sizeof(int *));
	for(int r=0; r<mtx.numrib; r++) {
		row_ranges[r] = (int *)malloc((blocksize+1) * sizeof(int));
		struct csr rib = *mtx.csrs[r];
		int nnz_per_thread = rib.nnz/blocksize;
		//go through each row, deciding whether or not to assign it
		int cur_thread = 0;
		int last_ind = 0;
		row_ranges[r][0] = 0;
		for(int row=0; row<rib.m; row++) {
			if(cur_thread >= blocksize) {
				continue;
			}
			else if(rib.rowsts[row]-last_ind > nnz_per_thread) {
				row_ranges[r][cur_thread+1] = row;
				last_ind = rib.rowsts[row];
				cur_thread++;
			}
		}
		// hopefully we wont have any leftover threads but if we do
		while(cur_thread < blocksize) {
			row_ranges[r][cur_thread+1] = rib.m-1;
			cur_thread++;
		}
		// just for good measure
		row_ranges[r][blocksize] = rib.m-1;
	}
	return row_ranges;
}

// examines the system and creates a RELL format from a supplied matrix market filename
struct sysinfo get_system_info() {
	// get num devices and min constant memory size and min warp size and initialize devices
	int numdevices;
	unsigned int sharedsize = INT_MAX;
	int warpsize = INT_MAX;
	hipDeviceProp_t prop;
	hipGetDeviceCount(&numdevices);
	for(int i=0; i<numdevices; i++) {
		hipSetDevice(i);
		hipFree(0);
		hipGetDeviceProperties(&prop, i);
		if(prop.sharedMemPerBlock < sharedsize) {
			sharedsize = prop.sharedMemPerBlock;
		}
		if(prop.warpSize < warpsize) {
			warpsize = prop.warpSize;
		}
	}
	struct sysinfo ret;
	ret.numdevices = numdevices;
	ret.sharedsize = sharedsize;
	ret.warpsize = warpsize;
	return ret;
}

// using balance array moves ribbons into devices, returns array listing last ribbon of each device
struct gpu_data move_data_to_devices(struct ricsr mtx, float ** slicedx, float * balance_arr, int ** row_ranges, int numdevices, int blocksize) {
	// generate ribbon assignments to devices
	int * devribs = (int *)malloc(sizeof(int)*(numdevices+1));
	devribs[0] = 0;
	float sum = 0.0f;
	for(int i=1; i<numdevices; i++) {
		sum += balance_arr[i];
		devribs[i] = (int)round(sum*(mtx.numrib));
	}
	devribs[numdevices] = mtx.numrib;

	//setup gpu data container
	struct gpu_data ret;
	ret.x_slices = (float **)malloc(mtx.numrib*sizeof(float *));
	ret.numrib = mtx.numrib;
	ret.devribs = devribs;
	ret.csrs = (struct csr **)malloc(mtx.numrib*sizeof(struct csr *));
	ret.m = mtx.m;
	ret.m_padded = (*mtx.csrs[0]).m_padded;
	ret.row_ranges = (int **)malloc(mtx.numrib * sizeof(int *));

	// in parallel copy x slices and ribbons to each device
	# pragma omp parallel num_threads(numdevices)
	{
	int devid = omp_get_thread_num();
	hipError_t cudaStat1, cudaStat2, cudaStat3, cudaStat4, cudaStat5;
	cudaStat1 = hipSetDevice(devid);
	if(! (cudaStat1 == hipSuccess)) {
		printf("ERROR: failed to set device to %d\n", devid);
		printf("%s : %s\n", hipGetErrorName(cudaStat1), hipGetErrorString(cudaStat1));
	}

	// for each ribbon to be assigned to this device
	for(int r=devribs[devid]; r<devribs[devid+1]; r++) {
		// initialize variables
		struct csr * ribcontainer = (struct csr *)malloc(sizeof(struct csr));
		int * cols;
		int * rowsts;
		float * vals;
		float * x_slice;
		int * rib_row_ranges;
		int m = (*mtx.csrs[r]).m;
		int n = (*mtx.csrs[r]).n;
		int nnz = (*mtx.csrs[r]).nnz;
		// allocate memory on gpu
		cudaStat1 = hipMalloc(&cols, nnz * sizeof(int));
		cudaStat2 = hipMalloc(&vals, nnz * sizeof(float));
		cudaStat3 = hipMalloc(&rowsts, (m+1) * sizeof(int));
		cudaStat4 = hipMalloc(&x_slice, mtx.ribwidth * sizeof(float));
		cudaStat5 = hipMalloc(&rib_row_ranges, (blocksize+1)*sizeof(int));
		if (!(cudaStat1 == hipSuccess && cudaStat2 == hipSuccess && cudaStat3 == hipSuccess && cudaStat4 == hipSuccess && cudaStat5 == hipSuccess)) {
			printf("ERROR: failed to allocate device memory for ribbon number %d on device %d\n", r, devid);
			break;
		}
		// copy data to gpu
		cudaStat1 = hipMemcpy(cols, (*mtx.csrs[r]).cols, nnz*sizeof(int), hipMemcpyHostToDevice);
		cudaStat2 = hipMemcpy(vals, (*mtx.csrs[r]).vals, nnz*sizeof(float), hipMemcpyHostToDevice);
		cudaStat3 = hipMemcpy(rowsts, (*mtx.csrs[r]).rowsts, (m+1)*sizeof(int), hipMemcpyHostToDevice);
		cudaStat4 = hipMemcpy(x_slice, slicedx[r], n * sizeof(float), hipMemcpyHostToDevice);
		cudaStat5 = hipMemcpy(rib_row_ranges, row_ranges[r], (blocksize+1) * sizeof(int), hipMemcpyHostToDevice);
		if (!(cudaStat1 == hipSuccess && cudaStat2 == hipSuccess && cudaStat3 == hipSuccess && cudaStat4 == hipSuccess && cudaStat5 == hipSuccess)) {
			printf("ERROR: failed to copy ribbon number %d to device %d\n got errors:\n %s\n%s\n%s\n", r, devid, hipGetErrorString(cudaStat1), hipGetErrorString(cudaStat2), hipGetErrorString(cudaStat3));
			break;
		}

		// assign to structures
		ribcontainer->m = m;
		ribcontainer->n = n;
		ribcontainer->nnz = (*mtx.csrs[r]).nnz;
		ribcontainer->cols = cols;
		ribcontainer->vals = vals;
		ribcontainer->rowsts = rowsts;
		ribcontainer->m_padded = (*mtx.csrs[0]).m_padded;

		ret.csrs[r] = ribcontainer;
		ret.x_slices[r] = x_slice;
		ret.row_ranges[r] = rib_row_ranges;
		ret.fullribwidth = mtx.ribwidth;
	}
	}
	return ret;
}

void ricsr_spmv(int numdevices, struct gpu_data dcontainer, int m, float * result) {
	hipError_t cudaStat1;
	// first lets allocate device memory for the intermediate ribbon results
	// should be a totalribbons-sized array full of pointers to m-sized arrays
	float * ribbon_results[dcontainer.numrib];
	cudaStat1 = hipDeviceSynchronize();
	for(int d=0; d<numdevices; d++) {
		cudaStat1 = hipSetDevice(d);
		if(cudaStat1 != hipSuccess)
			printf("ERROR: failed to set device to %d\n", d);
		for(int r=dcontainer.devribs[d]; r<dcontainer.devribs[d+1]; r++) {
			cudaStat1 = hipMalloc(&(ribbon_results[r]), dcontainer.m_padded*sizeof(float));
			if(cudaStat1 != hipSuccess) {
				printf("ERROR: filed to allocated memory on device %d for ribbon %d result array\n", d, r);
			}
		}
	}

	// parallel across devices, solve ribbons one at a time
	#pragma omp parallel num_threads(numdevices)
	{
		int m_padded = dcontainer.m_padded;
		int devid = omp_get_thread_num();
		hipSetDevice(devid);
		// for each ribbon
		for(int r=dcontainer.devribs[devid]; r<dcontainer.devribs[devid+1]; r++) {
			struct csr devrib = *(dcontainer.csrs[r]);
			int ribwidth = devrib.n;

			if(VERBOSE) {
//				printf("\nrowsts: %p\ncols: %p\nvals: %p\nx_slices[r]: %p\nribbon_results[r]: %p\n\n", devrib.rowsts, devrib. cols, devrib.vals, dcontainer.x_slices[r], ribbon_results[r]);
			}


			ComputeRibbon<<<1,BLOCKSIZE>>>(devrib.rowsts, devrib.cols, devrib.vals, dcontainer.x_slices[r], ribwidth, r, dcontainer.fullribwidth, dcontainer.row_ranges[r], ribbon_results[r]);
		}
		// now sum all the ribbons on this device
		// ROWS MUST BE PADDED TO BLOCK SIZE!!!!!!!
		if(!(dcontainer.m_padded%BLOCKSIZE==0)) {
			printf("WARNING: Number of rows not padded to block size. You MUST fix this!!! Leave now! Do it!\n");
		}
		for(int r=dcontainer.devribs[devid+1]-1; r>dcontainer.devribs[devid]; r--) {
			VVSum<<<m_padded/BLOCKSIZE, BLOCKSIZE>>>(ribbon_results[r-1], ribbon_results[r]);
		}
		// bring it back home
		cudaStat1 = hipDeviceSynchronize();
		#pragma omp barrier
	}

	// now we can combine our results across devices
	// do we have enough ribbons already allocated on device 0?
	if(dcontainer.devribs[1] >= numdevices) {
		for(int d=1; d<numdevices; d++) {
			cudaStat1 = hipMemcpy(ribbon_results[d], ribbon_results[dcontainer.devribs[d]], m*sizeof(float), hipMemcpyDeviceToDevice);
			if(cudaStat1 != hipSuccess) {
				printf("ERROR: Failed to copy device %d result to device 0\n", d);
			}
		}
		// ADD ALPHA, BETA HERE BY WRITING A DIFFERENT KERNEL FUNCTION
		for(int d=numdevices-1; d>0; d--) {
			int m_padded = dcontainer.m_padded;
			VVSum<<<m_padded/BLOCKSIZE, BLOCKSIZE>>>(ribbon_results[d-1], ribbon_results[d]);
		}
	}
	else {
		printf("ERROR: Device 1 did not have enough ribbon result arrays already allocated to move the other device results to. You should \
			change this else statement from this annoying message to an actual solution to this problem that involves reallocating some memory \
			on that device. Thank you.\n");
	}
	// finally copy our result to the desired place on host
	cudaStat1 = hipMemcpy(result, ribbon_results[0], m*sizeof(float), hipMemcpyDeviceToHost);
}

int free_csr_dev(struct csr csr_d) {
	hipFree(csr_d.cols);
	hipFree(csr_d.vals);
	hipFree(&csr_d);
	return 0;
}

int free_ricsr_dev(struct ricsr ricsr_d) {
	for(int r=0; r<ricsr_d.numrib; r++) {
		free_csr_dev(*ricsr_d.csrs[r]);
	}
	hipFree(ricsr_d.csrs);
	hipFree(&ricsr_d);
	return 0;
}

void execute_cusparse_spmv(struct coo arr_coo, float * x, float * res) {
	// cusparse setup
	hipError_t cudaStat1, cudaStat2, cudaStat3, cudaStat4, cudaStat5;
	hipsparseStatus_t status;
	hipsparseHandle_t handle = 0;
	hipsparseCreate(&handle);
	hipsparseMatDescr_t descriptor = 0;

	// allocate device memory for CSR array, y, and x
	int * d_csrRowPtr;
	int * d_colInds;
	float * d_vals;
	float * d_y;
	float * d_x;
	cudaStat1 = hipMalloc(&d_csrRowPtr, arr_coo.m*sizeof(int));
	cudaStat2 = hipMalloc(&d_colInds, arr_coo.nnz*sizeof(int));
	cudaStat3 = hipMalloc(&d_vals, arr_coo.nnz*sizeof(float));
	cudaStat4 = hipMalloc(&d_y, arr_coo.m*sizeof(float));
	cudaStat5 = hipMalloc(&d_x, arr_coo.n*sizeof(float));
	if(!((cudaStat1 == hipSuccess) && (cudaStat2 == hipSuccess) && (cudaStat3 == hipSuccess) && (cudaStat4 == hipSuccess) && (cudaStat5 == hipSuccess))) {
		printf("cuSPARSE: Device memory allocation failed, exiting\n");
		return;
	}

	float alpha = 1.0;
	float beta = 0.0;

	// grab metavars, convert array to csr and free coo
	int m = arr_coo.m;
	int n = arr_coo.n;
	int nnz = arr_coo.nnz;
	struct csr arr_csr = coo_to_csr(arr_coo);
	free_coo(arr_coo);

	// copy csr, y, and x to device
	cudaStat1 = hipMemcpy(d_csrRowPtr, arr_csr.rowsts, m*sizeof(int), hipMemcpyHostToDevice);
	cudaStat2 = hipMemcpy(d_colInds, arr_csr.cols, nnz*sizeof(int), hipMemcpyHostToDevice);
	cudaStat3 = hipMemcpy(d_vals, arr_csr.vals, nnz*sizeof(float), hipMemcpyHostToDevice);
	cudaStat4 = hipMemcpy(d_x, x, n*sizeof(float), hipMemcpyHostToDevice);
	if(!((cudaStat1 == hipSuccess) && (cudaStat2 == hipSuccess) && (cudaStat3 == hipSuccess) && (cudaStat4 == hipSuccess))) {
		printf("cuSPARSE: Initial memory copy failed, exiting\n");
		return;
	}

	// initialize cusparse
	status = hipsparseCreate(&handle);
	if(status != HIPSPARSE_STATUS_SUCCESS) {
		printf("cuSPARSE environment initialization failed, exiting\n");
		return;
	}
	hipsparseCreateMatDescr(&descriptor);
	if(status != HIPSPARSE_STATUS_SUCCESS) {
		printf("cuSPARSE matrix descriptor initialization failed, exiting\n");
		return;
	}
	hipsparseSetMatType(descriptor, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descriptor, HIPSPARSE_INDEX_BASE_ZERO);

	// execute spmv!
	status = hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, nnz, &alpha, descriptor, d_vals, d_csrRowPtr, d_colInds, d_x, &beta, d_y);
	if(status != HIPSPARSE_STATUS_SUCCESS) {
		printf("cuSPARSE SpMV function execution failed! exiting\n");
		return;
	}

	// copy result back
	cudaStat1 = hipMemcpy(res, d_y, m*sizeof(float), hipMemcpyDeviceToHost);
	if(cudaStat1 != hipSuccess) {
		printf("cuSPARSE: Error copy result back to host, exiting\n");
		return;
	}
}

// prints all messages with v = 0 and those with v = 1 if VERBOSE
void printlog(int v) {
	if(!(v && !VERBOSE)) {
		struct timespec wall;
		clock_gettime(CLOCK_REALTIME, &wall);
		float walldiff = (float)(wall.tv_sec - wallstart.tv_sec);
		walldiff += ((float)(wall.tv_nsec - wallstart.tv_nsec))/1000000000;
		float clockdiff = ((float)(clock()-cpustart))/CLOCKS_PER_SEC;
		printf("(W: %.4fs | P: %.4fs) : ", walldiff, clockdiff);
		printf(logbuf);
		printf("\n");
	}
}

int main(int argc, char * argv[]) {
	// ensure usage
	if(argc != 2) {
		printf("Usage: spmv [matrix market file]\n");
		return 1;
	}
	// initialize timing and message variables
	logbuf = (char *)malloc(512 * sizeof(char));
	clock_gettime(CLOCK_REALTIME, &wallstart);
	cpustart = clock();

	// read file
	snprintf(logbuf, 512, "reading file %s", argv[1]);
	printlog(1);
	struct coo arr_coo = real_mm_to_coo(argv[1]);
	struct sysinfo system_info = get_system_info();

	// convert to ricsr
	if (system_info.sharedsize/sizeof(float) != SHAREDMEM) {
		printf("ERROR: the shared memory size defined is incorrect! You must fix this!\n");
	}
	int ribwidth = SHAREDMEM;
	snprintf(logbuf, 512, "converting COO to RICSR");
	printlog(1);
	struct ricsr arr_ricsr = coo_to_ricsr(arr_coo, ribwidth, BLOCKSIZE);
	
	// generate and slice an x vector
	snprintf(logbuf, 512, "generating random x");
	printlog(1);
	float * x = gen_rand_x(arr_coo.n, 0.0, 2.0);
	snprintf(logbuf, 512, "slicing x vector");
	printlog(1);
	float ** slicedx = slice_x(x, arr_coo.n, ribwidth, 32);

	// move data to gpu
	float balance_arr[] = {1.0};
	snprintf(logbuf, 512, "building row_ranges");
	printlog(1);
	int ** row_ranges = build_row_ranges(arr_ricsr, BLOCKSIZE);

	snprintf(logbuf, 512, "moving data to gpu");
	printlog(1);
	struct gpu_data dcontainer = move_data_to_devices(arr_ricsr, slicedx, balance_arr, row_ranges, system_info.numdevices, BLOCKSIZE);

	// allocate result vectors
	float * cusparse_res = (float *)malloc(sizeof(float)*arr_coo.m);
	float * ricsr_res = (float *)malloc(sizeof(float)*round_val(arr_coo.m, BLOCKSIZE));

	// do the cusparse and store it there
	snprintf(logbuf, 512, "executing cuSPARSE matrix-vector multiplication");
	printlog(1);
	execute_cusparse_spmv(arr_coo, x, cusparse_res);

	hipError_t cudaStat1 = hipDeviceSynchronize();
	if(!cudaStat1 == hipSuccess) {
		int devid;
		hipGetDevice(&devid);
		printf("ERROR: failed to synchronize device %d after cuSPARSE\n", devid);
	}
	snprintf(logbuf, 512, "executing RICSR matrix-vector multiplication");
	printlog(1);
	ricsr_spmv(system_info.numdevices, dcontainer, dcontainer.m, ricsr_res);

	// free stuff
//	free_coo(arr_coo);

	// compare results
	snprintf(logbuf, 512, "comparing results");
	printlog(1);
	float maxerr = 0.01f;
	if (!(arrs_are_same(ricsr_res, cusparse_res, arr_coo.m, maxerr))) {
		snprintf(logbuf, 512, "cuSPARSE achieved a different result from RICSR");
		printlog(0);
	}
	else {
		snprintf(logbuf, 512, "RICSR algorithm result matches CUSPARSE within max error of %.2f%\n", 100*maxerr);
		printlog(0);
	}
	if(ribwidth % BLOCKSIZE != 0) {
		printf("ERROR: you didn't align your ribbon width to block size!!! Go fix it!\n");
	}
}

